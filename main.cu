#include "hip/hip_runtime.h"
#ifndef UNICODE
#define UNICODE
#endif


#include <GL/freeglut.h>
#include <conio.h>
#include <windows.h>

#include <chrono>
#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "lib/direct2d.h"

#pragma comment(lib, "d2d1")

#define WINDOW_HEIGHT 720
#define WINDOW_WIDTH 720

struct Transformation {
    float scale;
    float shift_x;
    float shift_y;
};

UINT8* PixelArray;
Transformation transformation = {1.0f, 0.0f, 0.0f};
RECT ClientRect;
gr::Direct2d graphics;
const int a = 0x41;
const int d = 0x44;
const int s = 0x53;
const int w = 0x57;

__device__ int Mandelbrot(float Re, float Im, float accuracy) {
    float temp_Re = Re;
    float temp_Im = Im;
    for (int i = 0; i < 10000 * accuracy + 1000; ++i) {
        float temp = temp_Re * temp_Re - temp_Im * temp_Im + Re;
        temp_Im = 2 * temp_Re * temp_Im + Im;
        temp_Re = temp;
        if (temp_Re * temp_Re + temp_Im * temp_Im > 4.0f) {
            return 0;
        }
    }
    return 1;
}

__global__ void GetPicture(UINT8* colors, int width, int height, Transformation transformation) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x = idx % width;
    float y = idx / width;
    idx *= 4;
    if (idx < width * height * 4) {
        float Re = (2.f * x - width) * transformation.scale / width + transformation.shift_x;
        float Im = (2.f * y - height) * transformation.scale / height + transformation.shift_y;
        float accuracy = log(transformation.scale) / log(0.5);
        accuracy *= accuracy;
        if (Mandelbrot(Re, Im, accuracy)) {
            colors[idx] = 0;
            colors[idx + 1] = 0;
            colors[idx + 2] = 255;
            colors[idx + 3] = 255;
        } else {
            colors[idx] = 0;
            colors[idx + 1] = 0;
            colors[idx + 2] = 0;
            colors[idx + 3] = 255;
        }
    }
}

void PrintMandelbrot(UINT8* dest, RECT client_rect) {
    int width = client_rect.right - client_rect.left;
    int height = client_rect.bottom - client_rect.top;
    int N = width * height;
    hipDeviceProp_t prop;
    cudaGetDeviceProperties_v2(&prop, 0);
    int L = prop.maxThreadsPerBlock;

    UINT8* cuda_color_array;
    hipMalloc(&cuda_color_array, N * 4);

    GetPicture<<<(N + L - 1) / L, L>>>(cuda_color_array, width, height, transformation);
    hipMemcpy(dest, cuda_color_array, N * 4, hipMemcpyDeviceToHost);
}

LRESULT CALLBACK WindowProc(HWND hwnd, UINT uMsg, WPARAM wParam, LPARAM lParam);

int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR pCmdLine, int nCmdShow) {
    // Register the window class.
    const wchar_t CLASS_NAME[] = L"Sample Window Class";

    WNDCLASS wc = {};

    wc.lpfnWndProc = WindowProc;
    wc.hInstance = hInstance;
    wc.lpszClassName = CLASS_NAME;

    RegisterClass(&wc);

    // Create the window.

    HWND hwnd = CreateWindowEx(
            0,                  // Optional window styles.
            CLASS_NAME,         // Window class
            L"Mandelbrot",      // Window text
            WS_OVERLAPPEDWINDOW,// Window style

            // Size and position
            CW_USEDEFAULT, CW_USEDEFAULT, WINDOW_WIDTH, WINDOW_HEIGHT,

            nullptr,  // Parent window
            nullptr,  // Menu
            hInstance,// Instance handle
            nullptr   // Additional application data
    );

    if (hwnd == nullptr) {

        return 0;
    }

    if (graphics.Initialize(hwnd)) {

        return 1;
    };

    ShowWindow(hwnd, nCmdShow);

    GetClientRect(hwnd, &ClientRect);
    PixelArray = new UINT8[(ClientRect.right - ClientRect.left) * (ClientRect.bottom - ClientRect.top) * 4];
    PrintMandelbrot(PixelArray, ClientRect);

    // Run the message loop.
    MSG msg = {};
    while (GetMessage(&msg, nullptr, 0, 0) > 0) {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
    }

    return 0;
}

LRESULT CALLBACK WindowProc(HWND hwnd, UINT uMsg, WPARAM wParam, LPARAM lParam) {
    switch (uMsg) {
        case WM_DESTROY:
            PostQuitMessage(0);
            return 0;

        case WM_PAINT: {
            graphics.BeginDraw();
            D2D1_PIXEL_FORMAT format = {DXGI_FORMAT_B8G8R8A8_UNORM, D2D1_ALPHA_MODE_IGNORE};
            D2D1_BITMAP_PROPERTIES prop = {format, 0, 0};
            ID2D1Bitmap* bitmap;
            PrintMandelbrot(PixelArray, ClientRect);
            HRESULT res = graphics.render_target->CreateBitmap(D2D1::SizeU(ClientRect.right - ClientRect.left, ClientRect.bottom - ClientRect.top),
                                                               PixelArray,
                                                               (ClientRect.right - ClientRect.left) * 4,
                                                               prop,
                                                               &bitmap);
            if (res == S_OK) {
                graphics.render_target->DrawBitmap(bitmap);
            }

            graphics.EndDraw();

            break;
        }

        case WM_KEYDOWN: {
            switch (LOWORD(wParam)) {
                case w: {
                    transformation.shift_y -= 1 * transformation.scale;
                    RedrawWindow(hwnd, &ClientRect, nullptr, RDW_INVALIDATE | RDW_UPDATENOW);
                    break;
                }

                case a: {
                    transformation.shift_x -= 1 * transformation.scale;
                    RedrawWindow(hwnd, &ClientRect, nullptr, RDW_INVALIDATE | RDW_UPDATENOW);
                    break;
                }

                case s: {
                    //todo why -+
                    transformation.shift_y += 1 * transformation.scale;
                    RedrawWindow(hwnd, &ClientRect, nullptr, RDW_INVALIDATE | RDW_UPDATENOW);
                    break;
                }

                case d: {
                    transformation.shift_x += 1 * transformation.scale;
                    RedrawWindow(hwnd, &ClientRect, nullptr, RDW_INVALIDATE | RDW_UPDATENOW);
                    break;
                }
            }

            break;
        }

        case WM_MOUSEWHEEL: {
            int delta = GET_WHEEL_DELTA_WPARAM(wParam);
            while (delta >= WHEEL_DELTA) {
                transformation.scale *= 0.5f;
                delta -= WHEEL_DELTA;
            }
            while (delta < 0) {
                transformation.scale /= 0.5f;
                delta += WHEEL_DELTA;
            }
            RedrawWindow(hwnd, &ClientRect, nullptr, RDW_INVALIDATE | RDW_UPDATENOW);

            break;
        }
        default: {

            break;
        }


            return 0;
    }

    return DefWindowProc(hwnd, uMsg, wParam, lParam);
}
