#include <d2d1.h>

#include "direct2d.h"


gr::Direct2d::Direct2d() : factory(nullptr),
                           render_target(nullptr) {
}

gr::Direct2d::~Direct2d() {
    if (factory) {
        factory->Release();
    }
    if (render_target) {
        render_target->Release();
    }
}

HRESULT gr::Direct2d::Initialize(HWND hwnd) {
    HRESULT res = D2D1CreateFactory(D2D1_FACTORY_TYPE_SINGLE_THREADED, &factory);
    if (res != S_OK) {

        return 1;
    }
    RECT client_rect;
    GetClientRect(hwnd, &client_rect);
    res = factory->CreateHwndRenderTarget(D2D1::RenderTargetProperties(),
                                          D2D1::HwndRenderTargetProperties(hwnd, D2D1::SizeU(client_rect.right -
                                                                                                     client_rect.left,
                                                                                             client_rect.bottom -
                                                                                                     client_rect.top)),
                                          &render_target);
    if (res != S_OK) {

        return 1;
    }

    return 0;
}

void gr::Direct2d::BeginDraw() {
    render_target->BeginDraw();
}

void gr::Direct2d::EndDraw() {
    render_target->EndDraw();
}
